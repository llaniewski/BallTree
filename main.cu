#include <vector>
#include <random>
#include <hip/hip_runtime.h>

#include "BallTree.h"
#include "BallTree.hpp"

namespace old {
	#undef BALLTREE_H
	#include "old/BallTree.h"
	#include "old/BallTree.hpp"
};

struct ball {
	double pos[3];
	double rad;
};

struct Balls {
	int n;
	ball* balls;
	inline long size() { return n; }
	inline __host__ __device__ double getPos(int i, int j) { return balls[i].pos[j]; }
	inline __host__ __device__ double getRad(int i) { return balls[i].rad; }
};


int main(int argn, char** argv) {
	hipEvent_t start1, stop1, start2, stop2;
	float milliseconds = 0;
	hipEventCreate(&start1);
	hipEventCreate(&stop1);
	hipEventCreate(&start2);
	hipEventCreate(&stop2);

    int n = 100000;
    int thr = 16;
	if (argn > 1) n = atoi(argv[1]);
	if (argn > 2) thr = atoi(argv[2]);
    Balls balls;
    std::uniform_real_distribution<double> pos_dist(0, 1);
    std::uniform_real_distribution<double> rad_dist(0.01, 0.1);
    std::default_random_engine random_engine;
	balls.balls = new ball[n];
	balls.n = n;
    for (int i=0;i<n;i++) {
        for (int j=0;j<3;j++) balls.balls[i].pos[j] = pos_dist(random_engine);
        balls.balls[i].rad = rad_dist(random_engine);
    }
    old::BallTree<Balls> oldtree;
    oldtree.balls = &balls;
    printf("building old ...\n");
    hipEventRecord(start1);
    oldtree.Build();
	hipEventRecord(stop1);
	printf("done\n");

	hipEventSynchronize(stop1);
	hipEventElapsedTime(&milliseconds, start1, stop1);
	printf("time:%f\n", milliseconds);

//    BallTree<Balls> tree;
//    tree.balls = &balls;

        size_t N = balls.size();
        int* nr = new int[N];
	size_t M = N*2-1;
        tr_elem* tree = new tr_elem[N*2-1];

	int * gnr; tr_elem* gtree; ball* gballs; Balls* ggballs;
	hipMalloc( &gnr,   sizeof(int) * N);
	hipMalloc( &gtree, sizeof(tr_elem) * M);
	hipMalloc( &gballs, sizeof(ball) * N);
	hipMalloc( &ggballs, sizeof(Balls));
	{
		Balls cgballs;
		cgballs.balls = gballs;
		cgballs.n = balls.n;
		hipMemcpy( gballs, balls.balls, sizeof(ball)*N, hipMemcpyHostToDevice  );
		hipMemcpy( ggballs, &cgballs, sizeof(Balls), hipMemcpyHostToDevice  );
	}


	printf("building gpu ...\n");
	hipEventRecord(start2);
	switch (thr) {
	case   2: buildgpu<   2 > <<< 1,   2 >>>( ggballs, gnr, gtree, N); break;
	case   4: buildgpu<   4 > <<< 1,   4 >>>( ggballs, gnr, gtree, N); break;
	case   8: buildgpu<   8 > <<< 1,   8 >>>( ggballs, gnr, gtree, N); break;
	case  16: buildgpu<  16 > <<< 1,  16 >>>( ggballs, gnr, gtree, N); break;
	case  32: buildgpu<  32 > <<< 1,  32 >>>( ggballs, gnr, gtree, N); break;
	case  64: buildgpu<  64 > <<< 1,  64 >>>( ggballs, gnr, gtree, N); break;
	case 128: buildgpu< 128 > <<< 1, 128 >>>( ggballs, gnr, gtree, N); break;
	default: printf("wrong number of threads %d\n", thr); return -1;
	}
	hipEventRecord(stop2);
	printf("done\n");

	hipEventSynchronize(stop2);
	hipEventElapsedTime(&milliseconds, start2, stop2);
	printf("time:%f\n", milliseconds);

	hipMemcpy( tree, gtree, sizeof(tr_elem)*M, hipMemcpyDeviceToHost );

	if (oldtree.size() != M ) {
		printf("Wrong sizes: old:%ld new:%ld\n", oldtree.size(), M);
		return -1;
	}
	for (int i=0; i<M; i++) {
		tr_elem el2 = tree[i];
		old::tr_elem el1 = oldtree.Tree()[i];
		bool sel = true;
		sel &= el1.flag  == el2.flag;
		sel &= el1.right == el2.right;
		sel &= el1.back  == el2.back;
		sel &= el1.a     == el2.a;
		sel &= el1.b     == el2.b;

		if (!sel) {
			printf("======================== Wrong ! ========================\n");
			printf("index : %d\n", i );
			printf("flag  : old:%d new:%d\n", (int) el1.flag  , (int) el2.flag  );
			printf("right : old:%d new:%d\n", (int) el1.right , (int) el2.right  );
			printf("back  : old:%d new:%d\n", (int) el1.back  , (int) el2.back  );
			printf("a     : old:%lf new:%lf diff: %lf\n", el1.a     , el2.a, el2.a - el1.a );
			printf("b     : old:%lf new:%lf diff: %lf\n", el1.b     , el2.b, el2.b - el1.b );
			printf("======================== Wrong ! ========================\n");
			return -1;
		}
	}
	
    return 0;
}
